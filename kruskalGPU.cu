#include "hip/hip_runtime.h"
#include <emmintrin.h>
#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>

int find(int);
int uni(int,int);

int getWeight(int array[],int row, int col, int n);
int setWeight(int* array[],int row, int col, int n, int value);

/*
int i,j,k,a,b,u,v,ne=1;
int min,mincost=0,parent[9];
*/
typedef struct graph{
	int* array;
	int numVert;
	int numEdges;	
} Graph;

struct timeval start, end;

void starttime() {
  gettimeofday( &start, 0 );
}

void endtime(const char* c) {
   gettimeofday( &end, 0 );
   double elapsed = ( end.tv_sec - start.tv_sec ) * 1000.0 + ( end.tv_usec - start.tv_usec ) / 1000.0;
   printf("%s: %f ms\n", c, elapsed); 
}

// GPU function to square root values
__global__ void parallelMergeSort(int* array) {
   //int element = blockIdx.x*blockDim.x + threadIdx.x;
   //if (element < N) a[element] = sqrt(a[element]);
}

void gpu(int array[], int n) {
	//char allocatedVertices[n];
   //int numThreads = 1024;
   //int numBlocks = N / 1024 + 1;

   //float* gpuA;
   //int* gpuArray;
   //hipMalloc(allocatedVertices, n*sizeof(char));
   //hipMalloc(gpuArray, n*n*sizeof(int));
   //hipMemcpy(gpuArray, array, n*n*sizeof(int), hipMemcpyHostToDevice);
   //gpu_sqrt<<<numBlocks, numThreads>>>(gpuA, N);
   //hipMemcpy(a, gpuA, N*sizeof(float), hipMemcpyDeviceToHost);
   //hipFree(&gpuA);
}

 /*
 int getWeight(int array[],int row, int col, int n){
	 return array[row*n + col];
 }
 
 void setWeight(int* array[],int row, int col, int n, int value){
	 (*array)[row*n + col] = value;
 }
 */
/*
void normal(int* array[], int n)
{	
	printf("\n\tImplementation of Kruskal's algorithm non parallelized\n");
	
	starttime();
	while(ne < n)
	{
		for(i=1,min=999;i<=n;i++)
		{
			for(j=1;j <= n;j++)
			{
				if(getWeight((*array),i,j,n) < min)
				{
					min=getWeight((*array),i,j,n);
					a=u=i;
					b=v=j;
				}
			}
		}
		u=find(u);
		v=find(v);
		if(uni(u,v))
		{
			printf("%d edge (%d,%d) =%d\n",ne++,a,b,min);
			mincost +=min;
		}
		setWeight(array,i,j,n,999);
		setWeight(array,j,i,n,999);
	}
	printf("\n\tMinimum cost = %d\n",mincost);
	
	endtime("CPU");
}

int find(int i)
{
	while(parent[i])
	i=parent[i];
	return i;
}

int uni(int i,int j)
{
	if(i!=j)
	{
		parent[j]=i;
		return 1;
	}
	return 0;
}
*/

/*
	This generates a graph randomly. The array goes in the format ||V1|V2|weight||V2|V3|weight||...||
*/
Graph* genGraph(int numEdges, int numVert,unsigned int seed, int maxWeight){
	int* array = (int*)malloc(sizeof(int)*numEdges*3);
	Graph *graph;
	graph = (Graph*)malloc(sizeof(Graph));
	
	graph -> array = array;
	graph -> numEdges = numEdges;
	graph -> numVert = numVert;
	
	int i,j;
	
	//generating seed
	srand(seed);
	
	//assuring a complete graph
	for(i = 0; i < numVert - 1; i++){
		array[(i*3) + 0] = i;
		array[(i*3) + 1] = i+1;
		array[(i*3) + 2] = rand() % maxWeight;
	}
	
	int firstVert, secondVert;
	
	//randomly inserting edges
	for( j = i; j < numEdges; j++){
		firstVert = rand() % numVert;
		array[(j*3) + 0] = firstVert;
		
		while((secondVert = rand() % numVert) == firstVert);
			
		array[(j*3) + 1] = secondVert;
		array[(j*3) + 2] = rand() % maxWeight;
	}
	
	return graph;
	
	
}
/*
	Printing the nodes of the graph as a test
*/
void printGraph(Graph* graph){
	int numEdges = graph -> numEdges;
	int i;
	
	int* array = (graph -> array);
	
	for(i =0; i < numEdges ;i++){
		printf("%d-%d: %d\n",array[(i*3) + 0],array[(i*3) + 1],array[(i*3) + 2]);
	}
}
int main()                                                                                                                                                                                  
{         
	time_t t;
	Graph* theGraph;
	theGraph = genGraph(8,5,(unsigned) time(&t),100);
	
	printGraph(theGraph);	
	
	//normal(&a,n);
	//gpu(a,n);

  return 0;
}

